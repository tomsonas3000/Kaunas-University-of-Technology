#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>

const int ARRAY_SIZE = 33;
const int NAME_SIZE = 15;
const int ARRAY_SIZE_RESULTS = ARRAY_SIZE * NAME_SIZE;


using namespace std;

struct Player {
	char Name[NAME_SIZE];
	int ShotsTaken = 0;
	double Ppg = 0.0;
};

__global__ void workWithData(Player* players, int* index, char* results);
__device__ char* my_strcpy(char *dest, const char *src, char pps);
__device__ int filterData(Player player, char* filteredPlayer);
vector<string> split(const string &s, char delim);

int main()
{
	string line;
	ifstream file("IFF-8-11_JorudasT_L3_2.txt");
	Player players[ARRAY_SIZE];
	if (file.is_open())
	{
		int i = 0;
		while (getline(file, line))
		{
			vector<string> s = split(line, ',');
			Player player = Player();
			strcpy(player.Name, s[0].c_str());
			player.ShotsTaken = stoi(s[1]);
			player.Ppg = stod(s[2]);
			players[i] = player;
			i++;
		}
		file.close();
	}
	else cout << "Unable to open file";

	char results[ARRAY_SIZE_RESULTS];
	int initialIndex = 0;

	int *device_index;
	Player *device_players;
	char *device_results;

	hipMalloc(&device_players, ARRAY_SIZE * sizeof(Player));
	hipMalloc(&device_results, ARRAY_SIZE_RESULTS * sizeof(char));
	hipMalloc(&device_index, sizeof(int));

	hipMemcpy(device_players, &players, ARRAY_SIZE * sizeof(Player), hipMemcpyHostToDevice);
	hipMemcpy(device_results, &results, ARRAY_SIZE_RESULTS * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(device_index, &initialIndex, sizeof(int), hipMemcpyHostToDevice);

	workWithData << <1,5 >> > (device_players, device_index, device_results);
	hipDeviceSynchronize();
	hipMemcpy(&initialIndex, device_index, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&results, device_results, ARRAY_SIZE_RESULTS * sizeof(char), hipMemcpyDeviceToHost);

	ofstream resultsFile;
	resultsFile.open("IFF-8-11_JorudasT_L3a_rez.txt");

	for (int i = 0; i < initialIndex; i++) {
		resultsFile << results[i];
	}
	file.close();
	cout << endl << "Isspausdinta " << initialIndex / NAME_SIZE << " elementai" << endl;

	hipFree(device_players);
	hipFree(device_results);
	hipFree(device_index);

    return 0;
}

vector<string> split(const string &s, char delim) {
	vector<string> result;
	stringstream ss(s);
	string item;

	while (getline(ss, item, delim)) {
		result.push_back(item);
	}

	return result;
}


__global__ void workWithData(Player *players, int *index, char *results) {
	int lengthOfArrayPart = ARRAY_SIZE / blockDim.x;
	int startIndex = lengthOfArrayPart * threadIdx.x;
	int endIndex;
	if (threadIdx.x == blockDim.x - 1) {
		endIndex = ARRAY_SIZE;
	}
	else {
		endIndex = lengthOfArrayPart * (threadIdx.x + 1);
	}
	for (int i = startIndex; i < endIndex; i++) {
		Player player = players[i];
		char filteredPlayer[NAME_SIZE];
		int check = filterData(player, filteredPlayer);
		if (check == 0) {
			int localIndex = atomicAdd(index, NAME_SIZE);
			for (int j = 0; j < NAME_SIZE; j++) {
				results[localIndex + j] = filteredPlayer[j];
			}
		}
		
	}
}

__device__ char* my_strcpy(char *dest, const char *src, char pps) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
	i--;
	dest[i++] = '-';
	dest[i++] = pps;
	for (int j = i; j < NAME_SIZE; j++) {
		dest[j] = ' ';
	}
	return dest;
}

__device__ int filterData(Player player, char* filteredPlayer) {
	int pps = player.Ppg / player.ShotsTaken;
	if (pps < 5) {
		return 1;
	}
	char ppsChar = '0' + pps;
	my_strcpy(filteredPlayer, player.Name, ppsChar);
	return 0;
}